#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>
#include <json/json.h>

__global__ void generate_data(int *ids, int *values, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        hiprandState state;
        hiprand_init(0, idx, 0, &state);
        ids[idx] = idx;
        values[idx] = hiprand(&state) % 100;
    }
}

int main() {
    const int n = 10;
    int ids[n], values[n];
    int *d_ids, *d_values;

    hipMalloc((void**)&d_ids, n * sizeof(int));
    hipMalloc((void**)&d_values, n * sizeof(int));

    generate_data<<<1, n>>>(d_ids, d_values, n);

    hipMemcpy(ids, d_ids, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values, d_values, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_ids);
    hipFree(d_values);

    Json::Value data(Json::arrayValue);
    for (int i = 0; i < n; ++i) {
        Json::Value item;
        item["id"] = ids[i];
        item["value"] = values[i];
        data.append(item);
    }

    std::ofstream file("data.json");
    file << data.toStyledString();
    file.close();

    return 0;
}